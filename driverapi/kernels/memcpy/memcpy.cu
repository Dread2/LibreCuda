
#include <hip/hip_runtime.h>
#include <cstdint>

#define MEMCPY_BLOCK_SIZE 256UL

extern "C" __global__ void memcpyKernelHighBW(uint32_t *dst, const uint32_t *src) {
    size_t idx = ((MEMCPY_BLOCK_SIZE * (blockIdx.y * gridDim.x + blockIdx.x)) << 2) + threadIdx.x;

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        dst[idx] = src[idx];
        idx += MEMCPY_BLOCK_SIZE;
    }
}

extern "C" __global__ void memcpyKernelLowLatency(uint32_t *dst, const uint32_t *src, size_t n) {
    size_t tid = MEMCPY_BLOCK_SIZE * blockIdx.x + threadIdx.x;
    if (tid < n) {
        dst[tid] = src[tid];
    }
}

extern "C" __global__ void memcpyKernelTrailing(uint8_t *dst, const uint8_t *src, size_t n) {
    size_t tid = MEMCPY_BLOCK_SIZE * blockIdx.x + threadIdx.x;
    if (tid < n) {
        dst[tid] = src[tid];
    }
}