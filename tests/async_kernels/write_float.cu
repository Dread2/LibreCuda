
#include <hip/hip_runtime.h>
extern "C" __global__ void write_float(float *dst, float *input) {
    double x = 0;
    int n = 100000000;
    for (int i = 0; i < n; i++) {
        x += 1.0;
    }
    x /= n;
    *dst = (float) x + (*input);
}