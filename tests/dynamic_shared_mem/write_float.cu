
#include <hip/hip_runtime.h>
extern "C" __global__ void write_float_sum(float *dst, short a, float b) {
    extern __shared__ float sharedData[];

    sharedData[1024] = (float) a;
    sharedData[1025] = (float) b;

    *dst = (sharedData[1024] + sharedData[1025]);
}