
#include <hip/hip_runtime.h>
extern "C" __global__ void write_float(float *dst, size_t n) {
    size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    dst[tid] = 1.0f;
}