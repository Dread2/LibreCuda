
#include <hip/hip_runtime.h>
extern "C" __global__ void write_float_ptr(float *dst, float *src) {
    *dst = *src;
}

extern "C" __global__ void write_float_value(float *dst, float value) {
    *dst = value;
}

__device__ int global_int = 0x69;

extern "C" __global__ void write_float_sum(float *dst, short a, float b) {
    *dst = (a + b + global_int);
}